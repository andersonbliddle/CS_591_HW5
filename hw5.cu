#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

// Kernel to compute the next generation
__global__ void next_generation_shared(int *grid, int *new_grid, int rows, int cols) {
    // Assuming a block size of 16x16
    __shared__ int shared_grid[18][18];  // 2 extra rows/columns for ghost cells

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int global_x = bx * blockDim.x + tx;
    int global_y = by * blockDim.y + ty;

    // Load main cell and its neighborhood into shared memory
    if (global_x < cols && global_y < rows) {
        // Load the main cell
        shared_grid[ty+1][tx+1] = grid[global_y * cols + global_x];

        // Load halo cells
        // Top row
        if (ty == 0) {
            if (global_y > 0)
                shared_grid[0][tx+1] = grid[(global_y-1) * cols + global_x];
            
            // Corner cells
            if (tx == 0 && global_x > 0 && global_y > 0)
                shared_grid[0][0] = grid[(global_y-1) * cols + (global_x-1)];
            
            if (tx == blockDim.x-1 && global_x < cols-1 && global_y > 0)
                shared_grid[0][tx+2] = grid[(global_y-1) * cols + (global_x+1)];
        }

        // Bottom row
        if (ty == blockDim.y-1) {
            if (global_y < rows-1)
                shared_grid[ty+2][tx+1] = grid[(global_y+1) * cols + global_x];
            
            // Corner cells
            if (tx == 0 && global_x > 0 && global_y < rows-1)
                shared_grid[ty+2][0] = grid[(global_y+1) * cols + (global_x-1)];
            
            if (tx == blockDim.x-1 && global_x < cols-1 && global_y < rows-1)
                shared_grid[ty+2][tx+2] = grid[(global_y+1) * cols + (global_x+1)];
        }

        // Left column
        if (tx == 0 && global_x > 0)
            shared_grid[ty+1][0] = grid[global_y * cols + (global_x-1)];

        // Right column
        if (tx == blockDim.x-1 && global_x < cols-1)
            shared_grid[ty+1][tx+2] = grid[global_y * cols + (global_x+1)];
    }

    // Synchronize to ensure all shared memory is loaded
    __syncthreads();

    // Compute next state
    if (global_x >= 1 && global_x < cols-1 && global_y >= 1 && global_y < rows-1) {
        // Count live neighbors using shared memory
        int neighbors = 
            shared_grid[ty][tx] + 
            shared_grid[ty][tx+1] + 
            shared_grid[ty][tx+2] +
            shared_grid[ty+1][tx] + 
            shared_grid[ty+1][tx+2] +
            shared_grid[ty+2][tx] + 
            shared_grid[ty+2][tx+1] + 
            shared_grid[ty+2][tx+2];

        // Apply Game of Life rules
        if (neighbors <= 1 || neighbors >= 4)
            new_grid[global_y * cols + global_x] = 0;  // Dies
        else if (neighbors == 3)
            new_grid[global_y * cols + global_x] = 1;  // Born
        else
            new_grid[global_y * cols + global_x] = shared_grid[ty+1][tx+1];  // Stays the same
    }
}

// Function to initialize the grid with random values
void initialize_grid(int *grid, int rows, int cols) {
    srand(42);  // Fixed seed for reproducibility
    for (int i = 1; i < rows - 1; i++) {
        for (int j = 1; j < cols - 1; j++) {
            grid[i * cols + j] = rand() % 2;
        }
    }
}

// Output function
void outputtofile(char *output_file, int *grid, int rows, int cols) {
    FILE *file = fopen(output_file, "w");
    for (int i = 1; i < rows - 1; i++) {
        for (int j = 1; j < cols - 1; j++) {
            fprintf(file, "%i ", grid[i * cols + j]);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

// Function to get the current time in seconds
double get_time() {
    struct timeval tval;
    gettimeofday(&tval, NULL);
    return (double)tval.tv_sec + (double)tval.tv_usec / 1000000.0;
}

// Main function
int main(int argc, char **argv) {
    if (argc != 6) {
        printf("Usage: %s <dimensions (int)> <max_generations (int)> <num_threads (int)> <stagnationcheck (boolean 1 or 0)> <output directory (string)>\n", argv[0]);
        exit(-1);
    }

    // Parse command line arguments
    int dimensions = atoi(argv[1]);
    int max_generations = atoi(argv[2]);
    int block_size = atoi(argv[3]);
    int stagnationcheck = atoi(argv[5]);
    // Boolean for turning on and off stagnation check

    int rows = dimensions + 2;  // Adding ghost rows
    int cols = dimensions + 2;

    size_t grid_size = rows * cols * sizeof(int);

    // Allocate memory for grids on host
    int *host_grid = (int *)malloc(grid_size);
    int *host_new_grid = (int *)malloc(grid_size);

    // Initialize the grid
    initialize_grid(host_grid, rows, cols);

    // Allocate memory for grids on device
    int *dev_grid, *dev_new_grid;
    hipMalloc((void **)&dev_grid, grid_size);
    hipMalloc((void **)&dev_new_grid, grid_size);

    // Copy initial grid to device
    hipMemcpy(dev_grid, host_grid, grid_size, hipMemcpyHostToDevice);

    // Set up block and grid dimensions
    dim3 block_dim(block_size, block_size);
    dim3 grid_dim((cols + block_size - 1) / block_size, (rows + block_size - 1) / block_size);

    // Main simulation loop
    for (int gen = 0; gen < max_generations; gen++) {
        next_generation_shared<<<grid_dim, block_dim>>>(dev_grid, dev_new_grid, rows, cols);

        // Swap grids
        int *temp = dev_grid;
        dev_grid = dev_new_grid;
        dev_new_grid = temp;

        // Optional: Check for stagnation (if enabled)
        if (stagnationcheck) {
            // Add stagnation check logic here if required.
        }
    }

    // Copy final grid back to host
    hipMemcpy(host_grid, dev_grid, grid_size, hipMemcpyDeviceToHost);

    // Output file and directory (format output_N_N_gen_threads.txt)
    char output_file[200];
    sprintf(output_file, "%s/output%s_%s_%s.txt", argv[4], argv[1], argv[2], argv[3]);
    outputtofile(output_file, host_grid, rows, cols);

    // Free memory on device
    hipFree(dev_grid);
    hipFree(dev_new_grid);

    // Free memory on host
    free(host_grid);
    free(host_new_grid);

    return 0;
}